#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
The Langevin thermostat with the BAOAB splitting:
[1] Leimkuhler, Benedict, and Charles Matthews. Applied Mathematics Research
    eXpress 2013.1 (2013): 34-56.
[2] Fass, Josh, et al. Entropy 20.5 (2018): 318.
------------------------------------------------------------------------------*/

#include "ensemble_bao.cuh"
#include "langevin_utilities.cuh"
#include "utilities/common.cuh"
#include <cstdlib>

Ensemble_BAO::Ensemble_BAO(int t, int fg, int N, double T, double Tc)
{
  type = t;
  fixed_group = fg;
  temperature = T;
  temperature_coupling = Tc;
  c1 = exp(-1.0 / temperature_coupling);
  c2 = sqrt((1 - c1 * c1) * K_B * T);
  curand_states.resize(N);
  int grid_size = (N - 1) / 128 + 1;
  initialize_curand_states<<<grid_size, 128>>>(curand_states.data(), N, rand());
  CUDA_CHECK_KERNEL
}

Ensemble_BAO::Ensemble_BAO(
  int t,
  int fg,
  int source_input,
  int sink_input,
  int source_size,
  int sink_size,
  int source_offset,
  int sink_offset,
  double T,
  double Tc,
  double dT)
{
  type = t;
  fixed_group = fg;
  temperature = T;
  temperature_coupling = Tc;
  delta_temperature = dT;
  source = source_input;
  sink = sink_input;
  N_source = source_size;
  N_sink = sink_size;
  offset_source = source_offset;
  offset_sink = sink_offset;
  c1 = exp(-1.0 / temperature_coupling);
  c2_source = sqrt((1 - c1 * c1) * K_B * (T + dT));
  c2_sink = sqrt((1 - c1 * c1) * K_B * (T - dT));
  curand_states_source.resize(N_source);
  curand_states_sink.resize(N_sink);
  int grid_size_source = (N_source - 1) / 128 + 1;
  int grid_size_sink = (N_sink - 1) / 128 + 1;
  initialize_curand_states<<<grid_size_source, 128>>>(
    curand_states_source.data(), N_source, rand());
  CUDA_CHECK_KERNEL
  initialize_curand_states<<<grid_size_sink, 128>>>(curand_states_sink.data(), N_sink, rand());
  CUDA_CHECK_KERNEL
  energy_transferred[0] = 0.0;
  energy_transferred[1] = 0.0;
}

Ensemble_BAO::~Ensemble_BAO(void)
{
  // nothing
}

// wrapper of the global Langevin thermostatting kernels
void Ensemble_BAO::integrate_nvt_lan(
  const GPU_Vector<double>& mass, GPU_Vector<double>& velocity_per_atom)
{
  const int number_of_atoms = mass.size();

  gpu_langevin<<<(number_of_atoms - 1) / 128 + 1, 128>>>(
    curand_states.data(),
    number_of_atoms,
    c1,
    c2,
    mass.data(),
    velocity_per_atom.data(),
    velocity_per_atom.data() + number_of_atoms,
    velocity_per_atom.data() + 2 * number_of_atoms);
  CUDA_CHECK_KERNEL

  gpu_find_momentum<<<4, 1024>>>(
    number_of_atoms,
    mass.data(),
    velocity_per_atom.data(),
    velocity_per_atom.data() + number_of_atoms,
    velocity_per_atom.data() + 2 * number_of_atoms);
  CUDA_CHECK_KERNEL

  gpu_correct_momentum<<<(number_of_atoms - 1) / 128 + 1, 128>>>(
    number_of_atoms,
    velocity_per_atom.data(),
    velocity_per_atom.data() + number_of_atoms,
    velocity_per_atom.data() + 2 * number_of_atoms);
  CUDA_CHECK_KERNEL
}

// wrapper of the local Langevin thermostatting kernels
void Ensemble_BAO::integrate_heat_lan(
  const std::vector<Group>& group,
  const GPU_Vector<double>& mass,
  GPU_Vector<double>& velocity_per_atom)
{
  const int number_of_atoms = mass.size();

  int Ng = group[0].number;

  std::vector<double> ek2(Ng);
  GPU_Vector<double> ke(Ng);

  find_ke<<<Ng, 512>>>(
    group[0].size.data(),
    group[0].size_sum.data(),
    group[0].contents.data(),
    mass.data(),
    velocity_per_atom.data(),
    velocity_per_atom.data() + number_of_atoms,
    velocity_per_atom.data() + 2 * number_of_atoms,
    ke.data());
  CUDA_CHECK_KERNEL

  ke.copy_to_host(ek2.data());
  energy_transferred[0] += ek2[source] * 0.5;
  energy_transferred[1] += ek2[sink] * 0.5;

  gpu_langevin<<<(N_source - 1) / 128 + 1, 128>>>(
    curand_states_source.data(),
    N_source,
    offset_source,
    group[0].contents.data(),
    c1,
    c2_source,
    mass.data(),
    velocity_per_atom.data(),
    velocity_per_atom.data() + number_of_atoms,
    velocity_per_atom.data() + 2 * number_of_atoms);
  CUDA_CHECK_KERNEL

  gpu_langevin<<<(N_sink - 1) / 128 + 1, 128>>>(
    curand_states_sink.data(),
    N_sink,
    offset_sink,
    group[0].contents.data(),
    c1,
    c2_sink,
    mass.data(),
    velocity_per_atom.data(),
    velocity_per_atom.data() + number_of_atoms,
    velocity_per_atom.data() + 2 * number_of_atoms);
  CUDA_CHECK_KERNEL

  find_ke<<<Ng, 512>>>(
    group[0].size.data(),
    group[0].size_sum.data(),
    group[0].contents.data(),
    mass.data(),
    velocity_per_atom.data(),
    velocity_per_atom.data() + number_of_atoms,
    velocity_per_atom.data() + 2 * number_of_atoms,
    ke.data());
  CUDA_CHECK_KERNEL

  ke.copy_to_host(ek2.data());
  energy_transferred[0] -= ek2[source] * 0.5;
  energy_transferred[1] -= ek2[sink] * 0.5;
}

// the A operator.
static __global__ void gpu_operator_A(
  const int number_of_particles,
  const int fixed_group,
  const int* group_id,
  const double g_time_step,
  const double* g_mass,
  double* g_x,
  double* g_y,
  double* g_z,
  double* g_vx,
  double* g_vy,
  double* g_vz,
  const double* g_fx,
  const double* g_fy,
  const double* g_fz)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < number_of_particles) {
    const double time_step = g_time_step;
    const double time_step_half = time_step * 0.5;
    double vx = g_vx[i];
    double vy = g_vy[i];
    double vz = g_vz[i];
    if (group_id[i] == fixed_group) {
      vx = 0.0;
      vy = 0.0;
      vz = 0.0;
    }
    g_x[i] += vx * time_step_half;
    g_y[i] += vy * time_step_half;
    g_z[i] += vz * time_step_half;
  }
}

static __global__ void gpu_operator_A(
  const int number_of_particles,
  const double g_time_step,
  const double* g_mass,
  double* g_x,
  double* g_y,
  double* g_z,
  double* g_vx,
  double* g_vy,
  double* g_vz,
  const double* g_fx,
  const double* g_fy,
  const double* g_fz)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < number_of_particles) {
    const double time_step = g_time_step;
    const double time_step_half = time_step * 0.5;
    double vx = g_vx[i];
    double vy = g_vy[i];
    double vz = g_vz[i];
    g_x[i] += vx * time_step_half;
    g_y[i] += vy * time_step_half;
    g_z[i] += vz * time_step_half;
  }
}

// wrapper of the above kernels
void Ensemble_BAO::operator_A(
  const double time_step,
  const std::vector<Group>& group,
  const GPU_Vector<double>& mass,
  const GPU_Vector<double>& force_per_atom,
  GPU_Vector<double>& position_per_atom,
  GPU_Vector<double>& velocity_per_atom)
{
  const int number_of_atoms = mass.size();

  if (fixed_group == -1) {
    gpu_operator_A<<<(number_of_atoms - 1) / 128 + 1, 128>>>(
      number_of_atoms,
      time_step,
      mass.data(),
      position_per_atom.data(),
      position_per_atom.data() + number_of_atoms,
      position_per_atom.data() + number_of_atoms * 2,
      velocity_per_atom.data(),
      velocity_per_atom.data() + number_of_atoms,
      velocity_per_atom.data() + 2 * number_of_atoms,
      force_per_atom.data(),
      force_per_atom.data() + number_of_atoms,
      force_per_atom.data() + 2 * number_of_atoms);
    CUDA_CHECK_KERNEL
  } else {
    gpu_operator_A<<<(number_of_atoms - 1) / 128 + 1, 128>>>(
      number_of_atoms,
      fixed_group,
      group[0].label.data(),
      time_step,
      mass.data(),
      position_per_atom.data(),
      position_per_atom.data() + number_of_atoms,
      position_per_atom.data() + number_of_atoms * 2,
      velocity_per_atom.data(),
      velocity_per_atom.data() + number_of_atoms,
      velocity_per_atom.data() + 2 * number_of_atoms,
      force_per_atom.data(),
      force_per_atom.data() + number_of_atoms,
      force_per_atom.data() + 2 * number_of_atoms);
    CUDA_CHECK_KERNEL
  }
}

// the B operator.
static __global__ void gpu_operator_B(
  const int number_of_particles,
  const int fixed_group,
  const int* group_id,
  const double g_time_step,
  const double* g_mass,
  double* g_x,
  double* g_y,
  double* g_z,
  double* g_vx,
  double* g_vy,
  double* g_vz,
  const double* g_fx,
  const double* g_fy,
  const double* g_fz)
{
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < number_of_particles) {
    const double time_step = g_time_step;
    const double time_step_half = time_step * 0.5;
    double vx = g_vx[i];
    double vy = g_vy[i];
    double vz = g_vz[i];
    const double mass_inv = 1.0 / g_mass[i];
    const double ax = g_fx[i] * mass_inv;
    const double ay = g_fy[i] * mass_inv;
    const double az = g_fz[i] * mass_inv;
    if (group_id[i] == fixed_group) {
      vx = 0.0;
      vy = 0.0;
      vz = 0.0;
    } else {
      vx += ax * time_step_half;
      vy += ay * time_step_half;
      vz += az * time_step_half;
    }
    g_vx[i] = vx;
    g_vy[i] = vy;
    g_vz[i] = vz;
  }
}

static __global__ void gpu_operator_B(
  const int number_of_particles,
  const double g_time_step,
  const double* g_mass,
  double* g_x,
  double* g_y,
  double* g_z,
  double* g_vx,
  double* g_vy,
  double* g_vz,
  const double* g_fx,
  const double* g_fy,
  const double* g_fz)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < number_of_particles) {
    const double time_step = g_time_step;
    const double time_step_half = time_step * 0.5;
    double vx = g_vx[i];
    double vy = g_vy[i];
    double vz = g_vz[i];
    const double mass_inv = 1.0 / g_mass[i];
    const double ax = g_fx[i] * mass_inv;
    const double ay = g_fy[i] * mass_inv;
    const double az = g_fz[i] * mass_inv;
    vx += ax * time_step_half;
    vy += ay * time_step_half;
    vz += az * time_step_half;
    g_vx[i] = vx;
    g_vy[i] = vy;
    g_vz[i] = vz;
  }
}

// wrapper of the above two kernels
void Ensemble_BAO::operator_B(
  const double time_step,
  const std::vector<Group>& group,
  const GPU_Vector<double>& mass,
  const GPU_Vector<double>& force_per_atom,
  GPU_Vector<double>& position_per_atom,
  GPU_Vector<double>& velocity_per_atom)
{
  const int number_of_atoms = mass.size();

  if (fixed_group == -1) {
    gpu_operator_B<<<(number_of_atoms - 1) / 128 + 1, 128>>>(
      number_of_atoms,
      time_step,
      mass.data(),
      position_per_atom.data(),
      position_per_atom.data() + number_of_atoms,
      position_per_atom.data() + number_of_atoms * 2,
      velocity_per_atom.data(),
      velocity_per_atom.data() + number_of_atoms,
      velocity_per_atom.data() + 2 * number_of_atoms,
      force_per_atom.data(),
      force_per_atom.data() + number_of_atoms,
      force_per_atom.data() + 2 * number_of_atoms);
  } else {
    gpu_operator_B<<<(number_of_atoms - 1) / 128 + 1, 128>>>(
      number_of_atoms,
      fixed_group,
      group[0].label.data(),
      time_step,
      mass.data(),
      position_per_atom.data(),
      position_per_atom.data() + number_of_atoms,
      position_per_atom.data() + number_of_atoms * 2,
      velocity_per_atom.data(),
      velocity_per_atom.data() + number_of_atoms,
      velocity_per_atom.data() + 2 * number_of_atoms,
      force_per_atom.data(),
      force_per_atom.data() + number_of_atoms,
      force_per_atom.data() + 2 * number_of_atoms);
  }
  CUDA_CHECK_KERNEL
}

void Ensemble_BAO::compute1(
  const double time_step,
  const std::vector<Group>& group,
  Box& box,
  Atom& atom,
  GPU_Vector<double>& thermo)
{
  if (type == 5) {
    operator_B(
      time_step,
      group,
      atom.mass,
      atom.force_per_atom,
      atom.position_per_atom,
      atom.velocity_per_atom);

    operator_A(
      time_step,
      group,
      atom.mass,
      atom.force_per_atom,
      atom.position_per_atom,
      atom.velocity_per_atom);

    integrate_nvt_lan(atom.mass, atom.velocity_per_atom);

    operator_A(
      time_step,
      group,
      atom.mass,
      atom.force_per_atom,
      atom.position_per_atom,
      atom.velocity_per_atom);
  } else {
    operator_B(
      time_step,
      group,
      atom.mass,
      atom.force_per_atom,
      atom.position_per_atom,
      atom.velocity_per_atom);

    operator_A(
      time_step,
      group,
      atom.mass,
      atom.force_per_atom,
      atom.position_per_atom,
      atom.velocity_per_atom);

    integrate_heat_lan(group, atom.mass, atom.velocity_per_atom);

    operator_A(
      time_step,
      group,
      atom.mass,
      atom.force_per_atom,
      atom.position_per_atom,
      atom.velocity_per_atom);
  }
}

void Ensemble_BAO::compute2(
  const double time_step,
  const std::vector<Group>& group,
  Box& box,
  Atom& atom,
  GPU_Vector<double>& thermo)
{
  if (type == 5) {
    operator_B(
      time_step,
      group,
      atom.mass,
      atom.force_per_atom,
      atom.position_per_atom,
      atom.velocity_per_atom);

    find_thermo(
      true,
      box.get_volume(),
      group,
      atom.mass,
      atom.potential_per_atom,
      atom.velocity_per_atom,
      atom.virial_per_atom,
      thermo);
  } else {
    operator_B(
      time_step,
      group,
      atom.mass,
      atom.force_per_atom,
      atom.position_per_atom,
      atom.velocity_per_atom);
  }
}