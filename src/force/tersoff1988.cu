#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
The version of the Tersoff potential as described in
    [1] J. Tersoff, New empirical approach for the structure and energy
    of covalent systems, PRB 37, 6991 (1988).
------------------------------------------------------------------------------*/

#include "neighbor.cuh"
#include "tersoff1988.cuh"
#include "utilities/common.cuh"
#include "utilities/error.cuh"
#include <vector>

#define LDG(a, n) __ldg(a + n)
#define BLOCK_SIZE_FORCE 64 // 128 is also good
#define EPSILON 1.0e-15

// Easy labels for indexing
#define A 0
#define B 1
#define LAMBDA 2
#define MU 3
#define BETA 4
#define EN 5 // special name for n to avoid conflict
#define C 6
#define D 7
#define H 8
#define R1 9
#define R2 10
#define M 11
#define ALPHA 12
#define GAMMA 13
#define C2 14
#define D2 15
#define ONE_PLUS_C2OVERD2 16
#define PI_FACTOR 17
#define MINUS_HALF_OVER_N 18

#define NUM_PARAMS 19

Tersoff1988::Tersoff1988(FILE* fid, int num_of_types, const int num_atoms)
{
  num_types = num_of_types;
  printf("Use Tersoff-1988 (%d-element) potential with element(s):", num_types);
  for (int n = 0; n < num_types; ++n) {
    char atom_symbol[10];
    int count = fscanf(fid, "%s", atom_symbol);
    PRINT_SCANF_ERROR(count, 1, "Reading error for Tersoff-1988 potential.");
    printf(" %s", atom_symbol);
  }
  printf("\n");

  int n_entries = num_types * num_types * num_types;
  // 14 parameters per entry of tersoff1988 + 5 pre-calculated values
  std::vector<double> cpu_ters(n_entries * NUM_PARAMS);

  char err[50] = "Error: Illegal Tersoff parameter.";
  rc = 0;
  int count;
  double a, b, lambda, mu, beta, n, c, d, h, r1, r2, m, alpha, gamma;
  for (int i = 0; i < n_entries; i++) {
    count = fscanf(
      fid,
      "%lf%lf%lf%lf%lf%lf%lf%lf%lf%lf%lf%lf%lf%lf",
      &a,
      &b,
      &lambda,
      &mu,
      &beta,
      &n,
      &c,
      &d,
      &h,
      &r1,
      &r2,
      &m,
      &alpha,
      &gamma);
    if (count != 14) {
      printf("Error: reading error for potential.in.\n");
      exit(1);
    }

    int m_int = round(m);
    // Parameter checking
    if (a < 0.0) {
      printf("%s A must be >= 0.\n", err);
      exit(1);
    }
    if (b < 0.0) {
      printf("%s B must be >= 0.\n", err);
      exit(1);
    }
    if (lambda < 0.0) {
      printf("%s Lambda must be >= 0.\n", err);
      exit(1);
    }
    if (mu < 0.0) {
      printf("%s Mu must be >= 0.\n", err);
      exit(1);
    }
    if (beta < 0.0) {
      printf("%s Beta must be >= 0.\n", err);
      exit(1);
    }
    if (n < 0.0) {
      printf("%s n must be >= 0.\n", err);
      exit(1);
    }
    if (c < 0.0) {
      printf("%s c must be >= 0.\n", err);
      exit(1);
    }
    if (d < 0.0) {
      printf("%s d must be >= 0.\n", err);
      exit(1);
    }
    if (r1 < 0.0) {
      printf("%s R must be >= 0.\n", err);
      exit(1);
    }
    if (r2 < 0.0) {
      printf("%s S must be >= 0.\n", err);
      exit(1);
    }
    if (r2 < r1) {
      printf("%s S-R must be >= 0.\n", err);
      exit(1);
    }
    if (m_int != 3 && m_int != 1) {
      printf("%s m must be 1 or 3.\n", err);
      exit(1);
    }
    if (gamma < 0.0) {
      printf("%s Gamma must be >= 0.\n", err);
      exit(1);
    }

    cpu_ters[i * NUM_PARAMS + A] = a;
    cpu_ters[i * NUM_PARAMS + B] = b;
    cpu_ters[i * NUM_PARAMS + LAMBDA] = lambda;
    cpu_ters[i * NUM_PARAMS + MU] = mu;
    cpu_ters[i * NUM_PARAMS + BETA] = beta;
    cpu_ters[i * NUM_PARAMS + EN] = n;
    cpu_ters[i * NUM_PARAMS + C] = c;
    cpu_ters[i * NUM_PARAMS + D] = d;
    cpu_ters[i * NUM_PARAMS + H] = h;
    cpu_ters[i * NUM_PARAMS + R1] = r1;
    cpu_ters[i * NUM_PARAMS + R2] = r2;
    cpu_ters[i * NUM_PARAMS + M] = m_int;
    if (alpha < EPSILON) {
      cpu_ters[i * NUM_PARAMS + ALPHA] = 0.0;
    } else {
      cpu_ters[i * NUM_PARAMS + ALPHA] = alpha;
    }
    cpu_ters[i * NUM_PARAMS + GAMMA] = gamma;
    cpu_ters[i * NUM_PARAMS + C2] = c * c;
    cpu_ters[i * NUM_PARAMS + D2] = d * d;
    cpu_ters[i * NUM_PARAMS + ONE_PLUS_C2OVERD2] =
      1.0 + cpu_ters[i * NUM_PARAMS + C2] / cpu_ters[i * NUM_PARAMS + D2];
    cpu_ters[i * NUM_PARAMS + PI_FACTOR] = PI / (r2 - r1);
    cpu_ters[i * NUM_PARAMS + MINUS_HALF_OVER_N] = -0.5 / n;
    rc = r2 > rc ? r2 : rc;
  }

  int num_of_neighbors = 50 * num_atoms;
  tersoff_data.b.resize(num_of_neighbors);
  tersoff_data.bp.resize(num_of_neighbors);
  tersoff_data.f12x.resize(num_of_neighbors);
  tersoff_data.f12y.resize(num_of_neighbors);
  tersoff_data.f12z.resize(num_of_neighbors);
  tersoff_data.NN.resize(num_atoms);
  tersoff_data.NL.resize(num_of_neighbors);
  tersoff_data.cell_count.resize(num_atoms);
  tersoff_data.cell_count_sum.resize(num_atoms);
  tersoff_data.cell_contents.resize(num_atoms);
  ters.resize(n_entries * NUM_PARAMS);
  ters.copy_from_host(cpu_ters.data());
}

Tersoff1988::~Tersoff1988(void)
{
  // nothing
}

static __device__ void
find_fr_and_frp(int i, const double* __restrict__ ters, double d12, double& fr, double& frp)
{
  fr = LDG(ters, i + A) * exp(-LDG(ters, i + LAMBDA) * d12);
  frp = -LDG(ters, i + LAMBDA) * fr;
}

static __device__ void
find_fa_and_fap(int i, const double* __restrict__ ters, double d12, double& fa, double& fap)
{
  fa = LDG(ters, i + B) * exp(-LDG(ters, i + MU) * d12);
  fap = -LDG(ters, i + MU) * fa;
}

static __device__ void find_fa(int i, const double* __restrict__ ters, double d12, double& fa)
{
  fa = LDG(ters, i + B) * exp(-LDG(ters, i + MU) * d12);
}

static __device__ void
find_fc_and_fcp(int i, const double* __restrict__ ters, double d12, double& fc, double& fcp)
{
  if (d12 < LDG(ters, i + R1)) {
    fc = 1.0;
    fcp = 0.0;
  } else if (d12 < LDG(ters, i + R2)) {
    fc = cos(LDG(ters, i + PI_FACTOR) * (d12 - LDG(ters, i + R1))) * 0.5 + 0.5;
    fcp =
      -sin(LDG(ters, i + PI_FACTOR) * (d12 - LDG(ters, i + R1))) * LDG(ters, i + PI_FACTOR) * 0.5;
  } else {
    fc = 0.0;
    fcp = 0.0;
  }
}

static __device__ void find_fc(int i, const double* __restrict__ ters, double d12, double& fc)
{
  if (d12 < LDG(ters, i + R1)) {
    fc = 1.0;
  } else if (d12 < LDG(ters, i + R2)) {
    fc = cos(LDG(ters, i + PI_FACTOR) * (d12 - LDG(ters, i + R1))) * 0.5 + 0.5;
  } else {
    fc = 0.0;
  }
}

static __device__ void
find_g_and_gp(int i, const double* __restrict__ ters, double cos, double& g, double& gp)
{
  double temp = LDG(ters, i + D2) + (cos - LDG(ters, i + H)) * (cos - LDG(ters, i + H));
  g = LDG(ters, i + GAMMA) * (LDG(ters, i + ONE_PLUS_C2OVERD2) - LDG(ters, i + C2) / temp);
  gp = LDG(ters, i + GAMMA) * (2.0 * LDG(ters, i + C2) * (cos - LDG(ters, i + H)) / (temp * temp));
}

static __device__ void find_g(int i, const double* __restrict__ ters, double cos, double& g)
{
  double temp = LDG(ters, i + D2) + (cos - LDG(ters, i + H)) * (cos - LDG(ters, i + H));
  g = LDG(ters, i + GAMMA) * (LDG(ters, i + ONE_PLUS_C2OVERD2) - LDG(ters, i + C2) / temp);
}

static __device__ void
find_e_and_ep(int i, const double* __restrict__ ters, double d12, double d13, double& e, double& ep)
{
  if (LDG(ters, i + ALPHA) < EPSILON) {
    e = 1.0;
    ep = 0.0;
  } else {
    double r = d12 - d13;
    if (LDG(ters, i + M) > 2.0) // if m == 3.0
    {
      e = exp(LDG(ters, i + ALPHA) * r * r * r);
      ep = LDG(ters, i + ALPHA) * 3.0 * r * r * e;
    } else {
      e = exp(LDG(ters, i + ALPHA) * r);
      ep = LDG(ters, i + ALPHA) * e;
    }
  }
}

static __device__ void
find_e(int i, const double* __restrict__ ters, double d12, double d13, double& e)
{
  if (LDG(ters, i + ALPHA) < EPSILON) {
    e = 1.0;
  } else {
    double r = d12 - d13;
    if (LDG(ters, i + M) > 2.0) {
      e = exp(LDG(ters, i + ALPHA) * r * r * r);
    } else {
      e = exp(LDG(ters, i + ALPHA) * r);
    }
  }
}

// step 1: pre-compute all the bond-order functions and their derivatives
static __global__ void find_force_tersoff_step1(
  const int number_of_particles,
  const int N1,
  const int N2,
  const Box box,
  const int num_types,
  const int* g_neighbor_number,
  const int* g_neighbor_list,
  const int* g_type,
  const double* __restrict__ ters,
  const double* __restrict__ g_x,
  const double* __restrict__ g_y,
  const double* __restrict__ g_z,
  double* g_b,
  double* g_bp)
{
  int num_types2 = num_types * num_types;
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
  if (n1 < N2) {
    int neighbor_number = g_neighbor_number[n1];
    int type1 = g_type[n1];
    double x1 = LDG(g_x, n1);
    double y1 = LDG(g_y, n1);
    double z1 = LDG(g_z, n1);
    for (int i1 = 0; i1 < neighbor_number; ++i1) {
      int n2 = g_neighbor_list[n1 + number_of_particles * i1];
      int type2 = g_type[n2];
      double x12 = g_x[n2] - x1;
      double y12 = g_y[n2] - y1;
      double z12 = g_z[n2] - z1;
      apply_mic(box, x12, y12, z12);
      double d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
      double zeta = 0.0;
      for (int i2 = 0; i2 < neighbor_number; ++i2) {
        int n3 = g_neighbor_list[n1 + number_of_particles * i2];
        if (n3 == n2) {
          continue;
        } // ensure that n3 != n2
        int type3 = g_type[n3];
        double x13 = g_x[n3] - x1;
        double y13 = g_y[n3] - y1;
        double z13 = g_z[n3] - z1;
        apply_mic(box, x13, y13, z13);
        double d13 = sqrt(x13 * x13 + y13 * y13 + z13 * z13);
        double cos123 = (x12 * x13 + y12 * y13 + z12 * z13) / (d12 * d13);
        double fc_ijk_13, g_ijk, e_ijk_12_13;
        int ijk = type1 * num_types2 + type2 * num_types + type3;
        if (d13 > LDG(ters, ijk * NUM_PARAMS + R2)) {
          continue;
        }
        find_fc(ijk * NUM_PARAMS, ters, d13, fc_ijk_13);
        find_g(ijk * NUM_PARAMS, ters, cos123, g_ijk);
        find_e(ijk * NUM_PARAMS, ters, d12, d13, e_ijk_12_13);
        zeta += fc_ijk_13 * g_ijk * e_ijk_12_13;
      }
      double bzn, b_ijj;
      int ijj = type1 * num_types2 + type2 * num_types + type2;
      bzn = pow(LDG(ters, ijj * NUM_PARAMS + BETA) * zeta, LDG(ters, ijj * NUM_PARAMS + EN));
      b_ijj = pow(1.0 + bzn, LDG(ters, ijj * NUM_PARAMS + MINUS_HALF_OVER_N));
      if (zeta < 1.0e-16) // avoid division by 0
      {
        g_b[i1 * number_of_particles + n1] = 1.0;
        g_bp[i1 * number_of_particles + n1] = 0.0;
      } else {
        g_b[i1 * number_of_particles + n1] = b_ijj;
        g_bp[i1 * number_of_particles + n1] = -b_ijj * bzn * 0.5 / ((1.0 + bzn) * zeta);
      }
    }
  }
}

// step 2: calculate all the partial forces dU_i/dr_ij
static __global__ void find_force_tersoff_step2(
  const int number_of_particles,
  const int N1,
  const int N2,
  Box box,
  const int num_types,
  const int* g_neighbor_number,
  const int* g_neighbor_list,
  const int* g_type,
  const double* __restrict__ ters,
  const double* __restrict__ g_b,
  const double* __restrict__ g_bp,
  const double* __restrict__ g_x,
  const double* __restrict__ g_y,
  const double* __restrict__ g_z,
  double* g_potential,
  double* g_f12x,
  double* g_f12y,
  double* g_f12z)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
  int num_types2 = num_types * num_types;
  if (n1 < N2) {
    int neighbor_number = g_neighbor_number[n1];
    int type1 = g_type[n1];
    double x1 = LDG(g_x, n1);
    double y1 = LDG(g_y, n1);
    double z1 = LDG(g_z, n1);
    double pot_energy = 0.0;
    for (int i1 = 0; i1 < neighbor_number; ++i1) {
      int index = i1 * number_of_particles + n1;
      int n2 = g_neighbor_list[index];
      int type2 = g_type[n2];

      double x12 = g_x[n2] - x1;
      double y12 = g_y[n2] - y1;
      double z12 = g_z[n2] - z1;
      apply_mic(box, x12, y12, z12);
      double d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
      double d12inv = 1.0 / d12;
      double fc_ijj_12, fcp_ijj_12;
      double fa_ijj_12, fap_ijj_12, fr_ijj_12, frp_ijj_12;
      int ijj = type1 * num_types2 + type2 * num_types + type2;
      find_fc_and_fcp(ijj * NUM_PARAMS, ters, d12, fc_ijj_12, fcp_ijj_12);
      find_fa_and_fap(ijj * NUM_PARAMS, ters, d12, fa_ijj_12, fap_ijj_12);
      find_fr_and_frp(ijj * NUM_PARAMS, ters, d12, fr_ijj_12, frp_ijj_12);

      // (i,j) part
      double b12 = LDG(g_b, index);
      double factor3 =
        (fcp_ijj_12 * (fr_ijj_12 - b12 * fa_ijj_12) + fc_ijj_12 * (frp_ijj_12 - b12 * fap_ijj_12)) *
        d12inv;
      double f12x = x12 * factor3 * 0.5;
      double f12y = y12 * factor3 * 0.5;
      double f12z = z12 * factor3 * 0.5;

      // accumulate potential energy
      pot_energy += fc_ijj_12 * (fr_ijj_12 - b12 * fa_ijj_12) * 0.5;

      // (i,j,k) part
      double bp12 = LDG(g_bp, index);
      for (int i2 = 0; i2 < neighbor_number; ++i2) {
        int index_2 = n1 + number_of_particles * i2;
        int n3 = g_neighbor_list[index_2];
        if (n3 == n2) {
          continue;
        }
        int type3 = g_type[n3];
        double x13 = g_x[n3] - x1;
        double y13 = g_y[n3] - y1;
        double z13 = g_z[n3] - z1;
        apply_mic(box, x13, y13, z13);
        double d13 = sqrt(x13 * x13 + y13 * y13 + z13 * z13);
        double fc_ikk_13, fc_ijk_13, fa_ikk_13, fc_ikj_12, fcp_ikj_12;
        int ikj = type1 * num_types2 + type3 * num_types + type2;
        int ikk = type1 * num_types2 + type3 * num_types + type3;
        int ijk = type1 * num_types2 + type2 * num_types + type3;
        find_fc(ikk * NUM_PARAMS, ters, d13, fc_ikk_13);
        find_fc(ijk * NUM_PARAMS, ters, d13, fc_ijk_13);
        find_fa(ikk * NUM_PARAMS, ters, d13, fa_ikk_13);
        find_fc_and_fcp(ikj * NUM_PARAMS, ters, d12, fc_ikj_12, fcp_ikj_12);
        double bp13 = LDG(g_bp, index_2);
        double one_over_d12d13 = 1.0 / (d12 * d13);
        double cos123 = (x12 * x13 + y12 * y13 + z12 * z13) * one_over_d12d13;
        double cos123_over_d12d12 = cos123 * d12inv * d12inv;
        double g_ijk, gp_ijk;
        find_g_and_gp(ijk * NUM_PARAMS, ters, cos123, g_ijk, gp_ijk);

        double g_ikj, gp_ikj;
        find_g_and_gp(ikj * NUM_PARAMS, ters, cos123, g_ikj, gp_ikj);

        // exp with d12 - d13
        double e_ijk_12_13, ep_ijk_12_13;
        find_e_and_ep(ijk * NUM_PARAMS, ters, d12, d13, e_ijk_12_13, ep_ijk_12_13);

        // exp with d13 - d12
        double e_ikj_13_12, ep_ikj_13_12;
        find_e_and_ep(ikj * NUM_PARAMS, ters, d13, d12, e_ikj_13_12, ep_ikj_13_12);

        // derivatives with cosine
        double dc = -fc_ijj_12 * bp12 * fa_ijj_12 * fc_ijk_13 * gp_ijk * e_ijk_12_13 +
                    -fc_ikj_12 * bp13 * fa_ikk_13 * fc_ikk_13 * gp_ikj * e_ikj_13_12;
        // derivatives with rij
        double dr = (-fc_ijj_12 * bp12 * fa_ijj_12 * fc_ijk_13 * g_ijk * ep_ijk_12_13 +
                     (-fcp_ikj_12 * bp13 * fa_ikk_13 * g_ikj * e_ikj_13_12 +
                      fc_ikj_12 * bp13 * fa_ikk_13 * g_ikj * ep_ikj_13_12) *
                       fc_ikk_13) *
                    d12inv;
        double cos_d = x13 * one_over_d12d13 - x12 * cos123_over_d12d12;
        f12x += (x12 * dr + dc * cos_d) * 0.5;
        cos_d = y13 * one_over_d12d13 - y12 * cos123_over_d12d12;
        f12y += (y12 * dr + dc * cos_d) * 0.5;
        cos_d = z13 * one_over_d12d13 - z12 * cos123_over_d12d12;
        f12z += (z12 * dr + dc * cos_d) * 0.5;
      }
      g_f12x[index] = f12x;
      g_f12y[index] = f12y;
      g_f12z[index] = f12z;
    }
    // save potential
    g_potential[n1] += pot_energy;
  }
}

// Wrapper of force evaluation for the Tersoff potential
void Tersoff1988::compute(
  Box& box,
  const GPU_Vector<int>& type,
  const GPU_Vector<double>& position_per_atom,
  GPU_Vector<double>& potential_per_atom,
  GPU_Vector<double>& force_per_atom,
  GPU_Vector<double>& virial_per_atom)
{
  const int number_of_atoms = type.size();
  int grid_size = (N2 - N1 - 1) / BLOCK_SIZE_FORCE + 1;

#ifdef USE_FIXED_NEIGHBOR
  static int num_calls = 0;
#endif
#ifdef USE_FIXED_NEIGHBOR
  if (num_calls++ == 0) {
#endif
    find_neighbor(
      N1,
      N2,
      rc,
      box,
      type,
      position_per_atom,
      tersoff_data.cell_count,
      tersoff_data.cell_count_sum,
      tersoff_data.cell_contents,
      tersoff_data.NN,
      tersoff_data.NL);
#ifdef USE_FIXED_NEIGHBOR
  }
#endif

  // pre-compute the bond order functions and their derivatives
  find_force_tersoff_step1<<<grid_size, BLOCK_SIZE_FORCE>>>(
    number_of_atoms,
    N1,
    N2,
    box,
    num_types,
    tersoff_data.NN.data(),
    tersoff_data.NL.data(),
    type.data(),
    ters.data(),
    position_per_atom.data(),
    position_per_atom.data() + number_of_atoms,
    position_per_atom.data() + number_of_atoms * 2,
    tersoff_data.b.data(),
    tersoff_data.bp.data());
  CUDA_CHECK_KERNEL

  // pre-compute the partial forces
  find_force_tersoff_step2<<<grid_size, BLOCK_SIZE_FORCE>>>(
    number_of_atoms,
    N1,
    N2,
    box,
    num_types,
    tersoff_data.NN.data(),
    tersoff_data.NL.data(),
    type.data(),
    ters.data(),
    tersoff_data.b.data(),
    tersoff_data.bp.data(),
    position_per_atom.data(),
    position_per_atom.data() + number_of_atoms,
    position_per_atom.data() + number_of_atoms * 2,
    potential_per_atom.data(),
    tersoff_data.f12x.data(),
    tersoff_data.f12y.data(),
    tersoff_data.f12z.data());
  CUDA_CHECK_KERNEL

  // the final step: calculate force and related quantities
  find_properties_many_body(
    box,
    tersoff_data.NN.data(),
    tersoff_data.NL.data(),
    tersoff_data.f12x.data(),
    tersoff_data.f12y.data(),
    tersoff_data.f12z.data(),
    position_per_atom,
    force_per_atom,
    virial_per_atom);
}
