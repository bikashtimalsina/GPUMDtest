/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
Some wrappers for the cuSOLVER library
------------------------------------------------------------------------------*/

#include "cusolver_wrapper.cuh"
#include "error.cuh"
#include "gpu_vector.cuh"
#include <hipsolver.h>
#include <vector>

void eig_hermitian_QR(size_t N, double* AR, double* AI, double* W_cpu)
{
  // get A
  size_t N2 = N * N;
  GPU_Vector<hipDoubleComplex> A(N2);
  std::vector<hipDoubleComplex> A_cpu(N2);

  for (size_t n = 0; n < N2; ++n) {
    A_cpu[n].x = AR[n];
    A_cpu[n].y = AI[n];
  }
  A.copy_from_host(A_cpu.data());

  // define W
  GPU_Vector<double> W(N);

  // get handle
  hipsolverHandle_t handle = NULL;
  hipsolverDnCreate(&handle);
  hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_NOVECTOR;
  hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

  // get work
  int lwork = 0;
  hipsolverDnZheevd_bufferSize(handle, jobz, uplo, N, A.data(), N, W.data(), &lwork);
  GPU_Vector<hipDoubleComplex> work(lwork);

  // get W
  GPU_Vector<int> info(1);
  hipsolverDnZheevd(handle, jobz, uplo, N, A.data(), N, W.data(), work.data(), lwork, info.data());
  W.copy_to_host(W_cpu);

  // free
  hipsolverDnDestroy(handle);
}

void eig_hermitian_Jacobi(size_t N, double* AR, double* AI, double* W_cpu)
{
  // get A
  size_t N2 = N * N;
  GPU_Vector<hipDoubleComplex> A(N2);
  std::vector<hipDoubleComplex> A_cpu(N2);
  for (size_t n = 0; n < N2; ++n) {
    A_cpu[n].x = AR[n];
    A_cpu[n].y = AI[n];
  }
  A.copy_from_host(A_cpu.data());

  // define W
  GPU_Vector<double> W(N);

  // get handle
  hipsolverHandle_t handle = NULL;
  hipsolverDnCreate(&handle);
  hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_NOVECTOR;
  hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

  // some parameters for the Jacobi method
  hipsolverSyevjInfo_t para = NULL;
  hipsolverDnCreateSyevjInfo(&para);

  // get work
  int lwork = 0;
  hipsolverDnZheevj_bufferSize(handle, jobz, uplo, N, A.data(), N, W.data(), &lwork, para);
  GPU_Vector<hipDoubleComplex> work(lwork);

  // get W
  GPU_Vector<int> info(1);
  hipsolverDnZheevj(
    handle, jobz, uplo, N, A.data(), N, W.data(), work.data(), lwork, info.data(), para);
  W.copy_to_host(W_cpu);

  // free
  hipsolverDnDestroy(handle);
  hipsolverDnDestroySyevjInfo(para);
}

void eigenvectors_symmetric_Jacobi(size_t N, double* A_cpu, double* W_cpu, double* eigenvectors_cpu)
{
  // get A
  size_t N2 = N * N;
  GPU_Vector<double> A(N2);
  A.copy_from_host(A_cpu);

  // define W
  GPU_Vector<double> W(N);

  // get handle
  hipsolverHandle_t handle = NULL;
  hipsolverDnCreate(&handle);
  hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR;
  hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

  // some parameters for the Jacobi method
  hipsolverSyevjInfo_t para = NULL;
  hipsolverDnCreateSyevjInfo(&para);

  // get work
  int lwork = 0;
  hipsolverDnDsyevj_bufferSize(handle, jobz, uplo, N, A.data(), N, W.data(), &lwork, para);
  GPU_Vector<double> work(lwork);

  // get W
  GPU_Vector<int> info(1);
  hipsolverDnDsyevj(
    handle, jobz, uplo, N, A.data(), N, W.data(), work.data(), lwork, info.data(), para);
  W.copy_to_host(W_cpu);
  A.copy_to_host(eigenvectors_cpu);

  // free
  hipsolverDnDestroy(handle);
  hipsolverDnDestroySyevjInfo(para);
}

void eig_hermitian_Jacobi_batch(size_t N, size_t batch_size, double* AR, double* AI, double* W_cpu)
{
  // get A
  size_t M = N * N * batch_size;
  GPU_Vector<hipDoubleComplex> A(M);
  std::vector<hipDoubleComplex> A_cpu(M);
  for (size_t n = 0; n < M; ++n) {
    A_cpu[n].x = AR[n];
    A_cpu[n].y = AI[n];
  }
  A.copy_from_host(A_cpu.data());

  // define W
  GPU_Vector<double> W(N * batch_size);

  // get handle
  hipsolverHandle_t handle = NULL;
  hipsolverDnCreate(&handle);
  hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_NOVECTOR;
  hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

  // some parameters for the Jacobi method
  hipsolverSyevjInfo_t para = NULL;
  hipsolverDnCreateSyevjInfo(&para);

  // get work
  int lwork = 0;
  hipsolverDnZheevjBatched_bufferSize(
    handle, jobz, uplo, N, A.data(), N, W.data(), &lwork, para, batch_size);
  GPU_Vector<hipDoubleComplex> work(lwork);

  // get W
  GPU_Vector<int> info(batch_size);
  hipsolverDnZheevjBatched(
    handle,
    jobz,
    uplo,
    N,
    A.data(),
    N,
    W.data(),
    work.data(),
    lwork,
    info.data(),
    para,
    batch_size);
  W.copy_to_host(W_cpu);

  // free
  hipsolverDnDestroy(handle);
  hipsolverDnDestroySyevjInfo(para);
}
